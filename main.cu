#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

using namespace std;
#include <stdio.h>
#include<chrono>
#include <ctime>
#include <cmath>
#include <iostream>
#include <fstream>
#include "functionsCpp.h"


__global__ void selectionSortCUDA(int* arr,int n)
{
    int ti = threadIdx.x;
    int tj = threadIdx.y;

    if (ti < n - 1) {
        int min_idx = ti;
        if ((tj>=ti+1) && (tj < n)) {
            if (arr[tj] < arr[min_idx]) {
                min_idx = tj;
            }

        }
        int temp = arr[min_idx];
        arr[min_idx] = arr[ti];
        arr[ti] = temp;
    }



}

//#include "hip/device_functions.h"
//#include <hip/hip_runtime.h>
/*
    nvcc main.cu -o out  -ccbin "C:\Program Files (x86)\Microsoft Visual Studio\2019\BuildTools\VC\Tools\MSVC\14.29.30133\bin\Hostx64\x64"
    start .\out.exe
 */


int main()
{
    system("chcp 65001"); //переключаем кодировку в кириллицу
    int choose = 0;
    cin>>choose;
    switch (choose)
    {
        case 1: {
            cout << "Введите размер массива: ";
            int n;
            cin >> n;
            int *mas = new int[n];
            for (int i = 0; i < n; i++) {
                cin >> mas[i];
            }
            selectionSort(mas, n);
            for(int i=0;i<n;i++)
            {
                cout<<mas[i]<<" ";
            }
            cout<<endl;
            break;
        }
        case 2: {
            cout<<"Сортировка простым выбором: "<<endl;
            for(int n1=pow(10,2);n1<=pow(10,6);n1*=10)
            {
                int *mas1 = new int[n1];
                for (int i = 0; i < n1; i++) {
                    mas1[i] = rand() % 100;
                }
                cout<<"Размер массива: "<<n1<<endl;
                //selectionSort(mas1, n1);
                //проверяем время работы
                auto start = chrono::high_resolution_clock::now();
                //запускаем функцию
                int *cudaA = 0;
                hipMalloc(&cudaA, sizeof(mas1));
                hipMemcpy(cudaA, mas1, sizeof(mas1), hipMemcpyHostToDevice);
                selectionSortCUDA << < 1, n1 >> > (cudaA, n1);
                hipMemcpy(mas1, cudaA, n1, hipMemcpyDeviceToHost);
                hipFree(cudaA);

                auto finish = chrono::high_resolution_clock::now();
                auto time = chrono::duration_cast<chrono::microseconds>(finish - start).count();
                cout<< "Время выполнения "<<n1<<" CUDA: "<<time<<"mks"<< endl;
                if(n1<=pow(10,6)){
                    int* masCPU = new int[n1];
                    for (int i = 0; i < n1; i++) {
                        masCPU[i] = rand() % 100;
                    }
                    auto startCPU = chrono::high_resolution_clock::now();

                    selectionSort(mas1, n1);

                    auto finishCPU = chrono::high_resolution_clock::now();
                    if(n1>=pow(10,4))
                    {
                        auto timeCPU = chrono::duration_cast<chrono::milliseconds>(finishCPU - startCPU).count();
                        cout << "Время выполнения " << n1 << " CPU: " << timeCPU << "ms" << endl;
                    }else{
                        auto timeCPU = chrono::duration_cast<chrono::microseconds>(finishCPU - startCPU).count();
                        cout<<"Время выполнения "<<n1<<" на CPU: "<<timeCPU<<"mks"<<endl;
                    }
                }
                delete[] mas1;
            }

            break;

        }
        default:
            cout<<"Неверный выбор";
            break;
    }

    system("pause");
    return 0;

}