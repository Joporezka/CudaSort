#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

using namespace std;
#include <stdio.h>
#include<chrono>
#include <ctime>
#include <cmath>
#include <iostream>
#include <fstream>

/*
    nvcc main.cu -o out  -ccbin "C:\Program Files (x86)\Microsoft Visual Studio\2019\BuildTools\VC\Tools\MSVC\14.29.30133\bin\Hostx64\x64"
    start .\out.exe
 */

__global__ void solve(int* mas,int n5)
{
    int i = threadIdx.x;

    long long int counter = 0, move = 0;

    for (int i1 = 0; i1 < n5; i1++)
    {
        counter++;
        int key = mas[i1];
        int j = i1 - 1;
        while (j >= 0 && key < mas[j]) {
            mas[j + 1] = mas[j];
            j--;
            move++;
        }
        mas[j + 1] = key;
    }



}

int main()
{
    system("chcp 65001"); //переключаем кодировку в кириллицу
    ofstream f("out.txt");

    const int n5 = 100000000;
    int* mas = new int[n5];


    if (n5 != 10)cout << "Проверка на случайно сгенерированном массиве размерности: " << n5;
    auto start_time = chrono::steady_clock::now();

    int* cudaA = 0;

    hipMalloc(&cudaA, sizeof(mas));

    hipMemcpy(cudaA, mas, sizeof(mas),hipMemcpyHostToDevice);



    for (int i = 0; i < n5; i++) {
        mas[i] = 1 + rand() % 100000;
    }



    solve << <1, n5 >> > (cudaA,n5);

    hipMemcpy(mas, cudaA, n5, hipMemcpyDeviceToHost);
    auto end_time = chrono::steady_clock::now();
    auto all_in_all = chrono::duration_cast<chrono::nanoseconds>(end_time - start_time);
    //cout << "\nC = " << counter << " M = " << move << " C+M = " << counter + move;
    f << "\n Time of compilation: " << double(all_in_all.count() / 1000000.0) << " ms\n\n";
    cout << "\n Time of compilation: " << double(all_in_all.count() / 1000000.0) << " ms\n\n";

    system("pause");
    return 0;

}